#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int dev = 0;
    hipSetDevice(dev);

    unsigned int size = 1 << 22;
    unsigned int bytes = size * sizeof(int);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device: %s\n", deviceProp.name);
    printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));

    // allocate host memory
    float *h_a = (float *)malloc(bytes);

    // allocate device memory
    float *d_a;
    hipMalloc((float **)&d_a, bytes);

    // initialize host memory
    for (unsigned int i = 0; i < size; i++) {
        h_a[i] = 0.1f;
    }
    
    // transfer data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

    // transfer data from device to host
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    
    // free memory
    hipFree(d_a);
    free(h_a);

    hipDeviceReset();
    return 0;
}