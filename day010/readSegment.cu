#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>

inline void CHECK(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("Error: %s:%d, ", file, line);
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

#define CHECK_CALL(call) CHECK((call), __FILE__, __LINE__)

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double EPSILON = 1.0E-8;
    bool match = 1;
    for (int i=0; i<N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > EPSILON) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match. \n\n");
}

void initialData (float *ip, int size) {
    // generate diferent seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost (float *A, float *B, float *C, const int N, int offset) {
    for (int idx=offset, k=0; idx<N; idx++, k++) {
        C[k] = A[idx] + B[idx];
    }
}

__global__ void warmup(float *A, float *B, float *C, const int N, int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i + offset;
    if (k < N) C[i] = A[k] + B[k];
}

__global__ void sumArraysOnGPUOffset(float *A, float *B, float *C, const int N, bool printDim, int offset) {

    if (printDim) {
        printf("threadIdx: (%d %d %d) blockIdx: (%d %d %d)  blockDim: (%d %d %d) gridDim: (%d %d %d)\n", 
        threadIdx.x, threadIdx.y, threadIdx.z,
        blockIdx.x, blockIdx.y, blockIdx.z,
        blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
    }

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i + offset;
    if (k < N) C[i] = A[k] + B[k];
}

void printArray(float *op, const int N) {
    for (int i=0; i<N; i++) {
        printf("%f ", op[i]);
    }
    printf("\n");
}


int main (int argc, char **argv) {
    printf("Starting %s...\n", argv[0]);

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 1 << 24;
    size_t nBytes = nElem * sizeof(float);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data on host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int blockSize = 512; // number of threads in a block
    dim3 block(blockSize);
    dim3 grid((nElem + block.x - 1) / block.x);

    bool printDim = false;
    int offset = 0;
    if (argc > 1) offset = atoi(argv[1]);

    double istart = cpuSecond();
    warmup <<<grid, block >>> (d_A, d_B, d_C, nElem, offset);
    hipDeviceSynchronize();
    double iElaps = cpuSecond() - istart;
    printf("Warmup configuration <<< %d, %d>>> with %d in %.5f seconds\n", grid.x, block.x, offset, iElaps);

    istart = cpuSecond();
    sumArraysOnGPUOffset <<<grid, block >>> (d_A, d_B, d_C, nElem, printDim, offset);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - istart;
    printf("Executed configuration <<< %d, %d>>> with %d in %.5f seconds\n", grid.x, block.x, offset, iElaps);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // printArray(hostRef, nElem);
    // printArray(gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    hipDeviceReset();
    return 0;
}