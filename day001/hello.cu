
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello, World from GPU! threadIdx: %d\n", threadIdx.x);
}

int main()
{
    printf("Hello, World from CPU!\n");
    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();
    return 0;
}