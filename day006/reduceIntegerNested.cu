#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return double(tp.tv_sec) + double(tp.tv_usec) / 1e6;
} 

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define FUN(a, b) ((a) + (b))

int recursiveReduce(int *A, int size){
    if (size == 1) return A[0];
    int stride = size / 2;
    for (int i = 0; i < stride; i++){
        A[i] = FUN(A[i], A[i + stride]);
    }
    return recursiveReduce(A, stride);
}

__global__ void gpuRecursiveReduce(int *g_idata, int *g_odata, unsigned int isize) {
    unsigned int tid = threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x; // offset to access each block's data
    int *odata = &g_odata[blockIdx.x];

    // stop condition
    if (isize == 2 && tid == 0) {
        g_odata[blockIdx.x] = FUN(idata[0], idata[1]);
        return;
    }

    // nested invocation
    int istride = isize >> 1;
    if (istride > 1 && tid < istride) {
        idata[tid] = FUN(idata[tid], idata[tid + istride]);
    }
    __syncthreads();

    if (tid == 0) {
        gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);

        // sync all child grids launched in this block
        hipDeviceSynchronize();
    }

    // sync at block level again
    __syncthreads();
}


int main(int argc, char **argv){

    int dev = 0;
    hipSetDevice(dev);

    bool bResult = false;

    // initialization
    int size = 1 << 24; // total number of elements to reduce
    printf("With array size %d\n", size);

    // execution configuration
    int blockSize = 64;
    if (argc > 1) {
        blockSize = atoi(argv[1]);
    }
    dim3 block(blockSize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x * sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for (int i = 0; i < size; i++){
        h_idata[i] = (int)(rand() & 0xFF); // 0 - 255
    }
    memcpy(tmp, h_idata, bytes);

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x * sizeof(int));

    // cpu reduction
    double iStart = cpuSecond();
    int cpuResult = recursiveReduce(tmp, size);
    double iElaps = cpuSecond() - iStart;
    printf("CPU reduce elapsed %f sec cpuResult: %d\n", iElaps, cpuResult);

    // kernel 5: gpuRecursiveReduce
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = cpuSecond();
    gpuRecursiveReduce<<<grid.x, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    int gpu_sum = 0;
    for (int i = 0; i < grid.x / 2; i++){
        gpu_sum = FUN(gpu_sum, h_odata[i]);
    }
    printf("gpuRecursiveReduce <<<%d, %d>>> elapsed %f sec result %d\n", grid.x, block.x, iElaps, gpu_sum);

    free(h_idata);
    free(h_odata);
    free(tmp);

    hipFree(d_idata);
    hipFree(d_odata);

    bResult = (gpu_sum == cpuResult);
    if (!bResult) printf("Test failed\n");
    return EXIT_SUCCESS;

}