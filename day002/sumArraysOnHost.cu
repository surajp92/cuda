#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

inline void CHECK(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("Error: %s:%d, ", file, line);
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

#define CHECK_CALL(call) CHECK((call), __FILE__, __LINE__)

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double EPSILON = 1.0E-8;
    bool match = 1;
    for (int i=0; i<N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > EPSILON) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match. \n\n");
}

void initialData (float *ip, int size) {
    // generate diferent seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost (float *A, float *B, float *C, const int N) {
    for (int idx=0; idx<N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N, bool printDim) {

    if (printDim) {
        printf("threadIdx: (%d %d %d) blockIdx: (%d %d %d)  blockDim: (%d %d %d) gridDim: (%d %d %d)\n", 
        threadIdx.x, threadIdx.y, threadIdx.z,
        blockIdx.x, blockIdx.y, blockIdx.z,
        blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
    }

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

void printArray(float *op, const int N) {
    for (int i=0; i<N; i++) {
        printf("%f ", op[i]);
    }
    printf("\n");
}


int main (int argc, char **argv) {
    printf("Starting %s...\n", argv[0]);

    // set up device
    int dev = 0;
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data on host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    int blockSize = 16; // number of threads in a block
    dim3 block(blockSize);
    dim3 grid((nElem + block.x - 1) / block.x);

    bool printDim = false;
    sumArraysOnGPU <<<grid, block >>> (d_A, d_B, d_C, nElem, printDim);
    printf("Executing configuration <<< %d, %d>>>\n", grid.x, block.x);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // printArray(h_A,nElem); 
    // printArray(h_B,nElem); 
    // printArray(hostRef,nElem); 
    
    return 0;
}